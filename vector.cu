
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vecAddKernel(float *A_d, float *B_d, float *C_d, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        C_d[i] = A_d[i] + B_d[i];
    }
}

void vecAdd(float *A_h, float *B_h, float *C_h, int n)
{
    float *A_d, *B_d, *C_d;

    int size = n * sizeof(float);
    hipMalloc((void **)&A_d, size);
    hipMalloc((void **)&B_d, size);
    hipMalloc((void **)&C_d, size);

    hipMemcpy(A_d, A_h, n, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, n, hipMemcpyHostToDevice);

    vecAddKernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, n, hipMemcpyDeviceToHost);
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    int n = 1000; // Example size
    float A_h[n], B_h[n], C_h[n];

    // Initialize A_h and B_h with some values
    for (int i = 0; i < n; i++)
    {
        A_h[i] = i * 1.0f;
        B_h[i] = i * 2.0f;
    }

    // Call the vector addition function
    vecAdd(A_h, B_h, C_h, n);

    // Print the result for verification
    for (int i = 0; i < 10; i++)
    {
        printf("C[%d] = %f\n", i, C_h[i]);
    }

    return 0;
}
